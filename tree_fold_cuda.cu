#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include <set>
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <unordered_map>
#include <fstream>
#include <string>
#include <map>
#include <cctype>
#include <sstream>
#include <cassert>
#include <cstdint>
#include <execution>
#include <functional>
#include <malloc.h>
#include <zstd.h>

// Error checking macro
#define CHECK_CUDA_ERROR(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

// Configuration parameters
#define MAX_ELEMENTS_PER_VECTOR 128
#define BLOCK_SIZE 256
#define TILE_SIZE_A 256  // Tile size for set A in tiled processing
#define TILE_SIZE_B 3072 // Tile size for set B in tiled processing
#define RESULTS_FLUSH_THRESHOLD 10000 // In-memory result limit before flushing to disk
#define CHUNK_SIZE (1024 * 4) // Number of items to load from a stream at a time

// --- Forward Declarations ---
typedef struct {
    int8_t* data;         // Flattened array of all elements
    int* offsets;      // Starting index for each vector/set
    int* sizes;        // Size of each vector/set
    int numItems;      // Number of vectors/sets
    int totalElements; // Total number of elements
    int8_t* deviceBuffer; // Reusable device buffer for operations
    int bufferSize;    // Size of the device buffer
} CudaSet;
struct LevelItem;
struct ProcessResult;
ProcessResult processPair_inMemory(const CudaSet& setA, const CudaSet& setB, int threshold, int level, bool verbose);
void processLargePair(const CudaSet& setA, const CudaSet& setB, int threshold, int level, bool verbose, 
                     std::unordered_map<size_t, std::vector<int>>& uniqueResults,
                     std::function<void()> flushCallback = nullptr);
ProcessResult processPair(const CudaSet& setA, const CudaSet& setB, int threshold, int level, bool verbose, bool allowStreaming);
LevelItem processStreamedPair(LevelItem& itemA, LevelItem& itemB, int threshold, int level, bool verbose);

// Absolute value functor for Thrust
struct AbsoluteFunctor {
    __host__ __device__
    int operator()(const int x) const {
        return x < 0 ? -x : x;
    }
};

// Simple JSON parser for our specific needs
class SimpleJsonParser {
private:
    std::string data;
    size_t pos = 0;

    void skipWhitespace() {
        while (pos < data.size() && std::isspace(data[pos])) pos++;
    }

    bool match(char c) {
        skipWhitespace();
        if (pos < data.size() && data[pos] == c) {
            pos++;
            return true;
        }
        return false;
    }

    std::string parseString() {
        if (!match('"')) return "";
        
        size_t start = pos;
        while (pos < data.size() && data[pos] != '"') {
            if (data[pos] == '\\' && pos + 1 < data.size()) pos++;
            pos++;
        }
        
        std::string result = data.substr(start, pos - start);
        match('"'); // Consume closing quote
        return result;
    }

    int parseInt() {
        skipWhitespace();
        
        bool negative = false;
        if (pos < data.size() && data[pos] == '-') {
            negative = true;
            pos++;
        }
        
        int value = 0;
        while (pos < data.size() && std::isdigit(data[pos])) {
            value = value * 10 + (data[pos] - '0');
            pos++;
        }
        
        return negative ? -value : value;
    }
    
    std::vector<int> parseArray() {
        std::vector<int> result;
        if (!match('[')) return result;
        
        while (!match(']')) {
            result.push_back(parseInt());
            match(','); // Consume comma if present
        }
        
        return result;
    }

public:
    SimpleJsonParser(const std::string& jsonData) : data(jsonData) {}
    
    struct Clause {
        std::string key;
        int condition_id1;
        int condition_id2;
        int consequence_id;
    };
    
    struct Matrix {
        std::string key;
        int rows;
        int cols;
        int type;
        std::vector<int> data;
    };
    
    std::vector<Clause> parseClauses() {
        std::vector<Clause> clauses;
        
        // Find "clauses": [ in the file
        size_t clausesStart = data.find("\"clauses\":");
        if (clausesStart == std::string::npos) return clauses;
        
        pos = clausesStart + 10; // Move past "clauses":
        skipWhitespace();
        
        if (!match('[')) return clauses;
        
        while (!match(']')) {
            if (!match('{')) break;
            
            Clause clause;
            
            while (!match('}')) {
                if (!match('"')) break;
                
                std::string key;
                while (pos < data.size() && data[pos] != '"') key += data[pos++];
                match('"');
                
                match(':');
                
                if (key == "key") {
                    clause.key = parseString();
                } else if (key == "condition_id1") {
                    clause.condition_id1 = parseInt();
                } else if (key == "condition_id2") {
                    clause.condition_id2 = parseInt();
                } else if (key == "consequence_id") {
                    clause.consequence_id = parseInt();
                } else {
                    // Skip unknown field
                    while (pos < data.size() && data[pos] != ',' && data[pos] != '}') pos++;
                }
                
                match(','); // Consume comma if present
            }
            
            clauses.push_back(clause);
            match(','); // Consume comma if present
        }
        
        return clauses;
    }
    
    std::vector<Matrix> parseMatrices() {
        std::vector<Matrix> matrices;
        
        // Find "matrices": [ in the file
        size_t matricesStart = data.find("\"matrices\":");
        if (matricesStart == std::string::npos) return matrices;
        
        pos = matricesStart + 11; // Move past "matrices":
        skipWhitespace();
        
        if (!match('[')) return matrices;
        
        while (!match(']')) {
            if (!match('{')) break;
            
            Matrix matrix;
            
            while (!match('}')) {
                if (!match('"')) break;
                
                std::string key;
                while (pos < data.size() && data[pos] != '"') key += data[pos++];
                match('"');
                
                match(':');
                
                if (key == "key") {
                    matrix.key = parseString();
                } else if (key == "rows") {
                    matrix.rows = parseInt();
                } else if (key == "cols") {
                    matrix.cols = parseInt();
                } else if (key == "type") {
                    matrix.type = parseInt();
                } else if (key == "data") {
                    matrix.data = parseArray();
                } else {
                    // Skip unknown field
                    while (pos < data.size() && data[pos] != ',' && data[pos] != '}') pos++;
                }
                
                match(','); // Consume comma if present
            }
            
            matrices.push_back(matrix);
            match(','); // Consume comma if present
        }
        
        return matrices;
    }
};

//-------------------------------------------------------------------------
// Host-side data structures
//-------------------------------------------------------------------------
typedef struct {
    std::vector<std::vector<int>> vectors;  // Original vectors 
} HostSet;



// Result buffer for parallel combination processing
typedef struct {
    int* data;         // Buffer for all potential results
    int* validFlags;   // Flags indicating if each combination is valid
    int* sizes;        // Size of each result set
    int maxResultSize; // Maximum possible size of a result
    int numCombinations; // Total number of combinations
} CombinationResultBuffer;

// Result struct to handle in-memory or streamed results
struct ProcessResult {
    CudaSet set;
    std::string streamPath; // Path to file if results are streamed
    int fromIdA = -1;
    int fromIdB = -1;
    size_t numResultItems = 0;
};


// Allocate memory for a CUDA set with additional buffer space
CudaSet allocateCudaSet(int numItems, int totalElements, int bufferSize = 0) {
    CudaSet set;
    set.numItems = numItems;
    set.totalElements = totalElements;
    
    CHECK_CUDA_ERROR(hipMalloc(&set.data, totalElements * sizeof(int8_t)));
    CHECK_CUDA_ERROR(hipMalloc(&set.offsets, numItems * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&set.sizes, numItems * sizeof(int)));
    
    // Allocate device buffer if size is specified
    if (bufferSize > 0) {
        CHECK_CUDA_ERROR(hipMalloc(&set.deviceBuffer, bufferSize * sizeof(int8_t)));
        set.bufferSize = bufferSize;
    } else {
        set.deviceBuffer = nullptr;
        set.bufferSize = 0;
    }
    
    return set;
}

// Free memory for a CUDA set
void freeCudaSet(CudaSet* set) {
    if (set->data) hipFree(set->data);
    if (set->offsets) hipFree(set->offsets);
    if (set->sizes) hipFree(set->sizes);
    if (set->deviceBuffer) hipFree(set->deviceBuffer);
    set->numItems = 0;
    set->totalElements = 0;
    set->bufferSize = 0;
    set->data = nullptr;
    set->offsets = nullptr;
    set->sizes = nullptr;
    set->deviceBuffer = nullptr;
}

// Allocate result buffer for parallel combination processing
CombinationResultBuffer allocateCombinationResultBuffer(int numItemsA, int numItemsB, int maxElementsPerVector) {
    CombinationResultBuffer buffer;
    buffer.numCombinations = numItemsA * numItemsB;
    buffer.maxResultSize = 2 * maxElementsPerVector; // Worst case: all elements from both vectors
    
    CHECK_CUDA_ERROR(hipMalloc(&buffer.data, buffer.numCombinations * buffer.maxResultSize * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&buffer.validFlags, buffer.numCombinations * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&buffer.sizes, buffer.numCombinations * sizeof(int)));
    
    // Initialize all valid flags to 0 (invalid)
    CHECK_CUDA_ERROR(hipMemset(buffer.validFlags, 0, buffer.numCombinations * sizeof(int)));
    
    return buffer;
}

// Free result buffer
void freeCombinationResultBuffer(CombinationResultBuffer* buffer) {
    if (buffer->data) hipFree(buffer->data);
    if (buffer->validFlags) hipFree(buffer->validFlags);
    if (buffer->sizes) hipFree(buffer->sizes);
    buffer->data = nullptr;
    buffer->validFlags = nullptr;
    buffer->sizes = nullptr;
}

// Host to device copy for a set (optimized to use pinned memory for larger transfers)
void copyHostToDevice(const HostSet& hostSet, CudaSet* cudaSet) {
    int numItems = hostSet.vectors.size();
    
    // Prepare host side arrays
    std::vector<int> hostIntData;
    std::vector<int> hostOffsets(numItems);
    std::vector<int> hostSizes(numItems);
    
    int currentOffset = 0;
    for (int i = 0; i < numItems; i++) {
        hostOffsets[i] = currentOffset;
        hostSizes[i] = hostSet.vectors[i].size();
        
        for (int j = 0; j < hostSet.vectors[i].size(); j++) {
            hostIntData.push_back(hostSet.vectors[i][j]);
        }
        
        currentOffset += hostSet.vectors[i].size();
    }
    
    // Convert to int8_t for device storage
    std::vector<int8_t> hostData(hostIntData.size());
    for (size_t i = 0; i < hostIntData.size(); ++i) {
        assert(hostIntData[i] >= INT8_MIN && hostIntData[i] <= INT8_MAX && "Input data exceeds int8_t range!");
        hostData[i] = static_cast<int8_t>(hostIntData[i]);
    }

    // Use pinned memory for large transfers
    int totalElements = hostData.size();
    int8_t* pinnedData = nullptr;
    int* pinnedOffsets = nullptr;
    int* pinnedSizes = nullptr;
    
    if (totalElements > 1024) {
        CHECK_CUDA_ERROR(hipHostMalloc((void**)&pinnedData, totalElements * sizeof(int8_t)));
        CHECK_CUDA_ERROR(hipHostMalloc(&pinnedOffsets, numItems * sizeof(int)));
        CHECK_CUDA_ERROR(hipHostMalloc(&pinnedSizes, numItems * sizeof(int)));
        
        memcpy(pinnedData, hostData.data(), totalElements * sizeof(int8_t));
        memcpy(pinnedOffsets, hostOffsets.data(), numItems * sizeof(int));
        memcpy(pinnedSizes, hostSizes.data(), numItems * sizeof(int));
    }
    
    // Allocate device memory
    *cudaSet = allocateCudaSet(numItems, totalElements, totalElements * 2);
    
    // Copy data to device
    if (totalElements > 1024) {
        CHECK_CUDA_ERROR(hipMemcpy(cudaSet->data, pinnedData, totalElements * sizeof(int8_t), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(cudaSet->offsets, pinnedOffsets, numItems * sizeof(int), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(cudaSet->sizes, pinnedSizes, numItems * sizeof(int), hipMemcpyHostToDevice));
        
        hipHostFree(pinnedData);
        hipHostFree(pinnedOffsets);
        hipHostFree(pinnedSizes);
    } else {
        CHECK_CUDA_ERROR(hipMemcpy(cudaSet->data, hostData.data(), totalElements * sizeof(int8_t), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(cudaSet->offsets, hostOffsets.data(), numItems * sizeof(int), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(cudaSet->sizes, hostSizes.data(), numItems * sizeof(int), hipMemcpyHostToDevice));
    }
}

// Device to host copy (optimized with streams for larger data)
HostSet copyDeviceToHost(const CudaSet& cudaSet) {
    HostSet hostSet;
    
    // Copy offsets and sizes
    std::vector<int> hostOffsets(cudaSet.numItems);
    std::vector<int> hostSizes(cudaSet.numItems);
    
    CHECK_CUDA_ERROR(hipMemcpy(hostOffsets.data(), cudaSet.offsets, cudaSet.numItems * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(hostSizes.data(), cudaSet.sizes, cudaSet.numItems * sizeof(int), hipMemcpyDeviceToHost));
    
    // For large data, use async transfers with streams
    std::vector<int8_t> hostData8(cudaSet.totalElements);
    
    if (cudaSet.totalElements > 1024) {
        hipStream_t stream;
        CHECK_CUDA_ERROR(hipStreamCreate(&stream));
        
        int8_t* pinnedData;
        CHECK_CUDA_ERROR(hipHostMalloc((void**)&pinnedData, cudaSet.totalElements * sizeof(int8_t)));
        
        CHECK_CUDA_ERROR(hipMemcpyAsync(pinnedData, cudaSet.data, cudaSet.totalElements * sizeof(int8_t), 
                                       hipMemcpyDeviceToHost, stream));
        CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
        
        memcpy(hostData8.data(), pinnedData, cudaSet.totalElements * sizeof(int8_t));
        
        hipHostFree(pinnedData);
        hipStreamDestroy(stream);
    } else {
        CHECK_CUDA_ERROR(hipMemcpy(hostData8.data(), cudaSet.data, cudaSet.totalElements * sizeof(int8_t), 
                                  hipMemcpyDeviceToHost));
    }
    
    // Reconstruct vectors
    hostSet.vectors.resize(cudaSet.numItems);
    
    // Convert back to int
    std::vector<int> hostData(cudaSet.totalElements);
    for (size_t i = 0; i < hostData8.size(); ++i) {
        hostData[i] = hostData8[i];
    }

    for (int i = 0; i < cudaSet.numItems; i++) {
        int offset = hostOffsets[i];
        int size = hostSizes[i];
        
        hostSet.vectors[i].resize(size);
        for (int j = 0; j < size; j++) {
            hostSet.vectors[i][j] = hostData[offset + j];
        }
    }
    
    return hostSet;
}

// Helper function to create a test set
HostSet createTestSet(const std::vector<std::vector<int>>& vectors) {
    HostSet set;
    set.vectors = vectors;
    return set;
}

//-------------------------------------------------------------------------
// CUDA Kernels and Device Functions
//-------------------------------------------------------------------------

// Device function to check if an element is in a set
__device__ bool deviceContains(const int* array, int size, int value) {
    for (int i = 0; i < size; i++) {
        if (array[i] == value) {
            return true;
        }
    }
    return false;
}

// Kernel to convert vector elements to unique elements (for Level 1 carry-over)
__global__ void convertToUniqueKernel(
    int8_t* inputData, int* inputOffsets, int* inputSizes, int numItems,
    int8_t* outputData, int* outputOffsets, int* outputSizes, int maxOutputSize
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= numItems) {
        return;
    }
    
    int inputOffset = inputOffsets[idx];
    int inputSize = inputSizes[idx];
    int outputOffset = outputOffsets[idx];
    
    // Local working memory for unique elements
    int localSet[MAX_ELEMENTS_PER_VECTOR];
    int localSetSize = 0;
    
    // Get unique elements
    for (int i = 0; i < inputSize; i++) {
        int val = inputData[inputOffset + i];
        if (!deviceContains(localSet, localSetSize, val)) {
            localSet[localSetSize++] = val;
        }
    }
    
    // Copy result to output
    outputSizes[idx] = localSetSize;
    for (int i = 0; i < localSetSize; i++) {
        outputData[outputOffset + i] = localSet[i];
    }
}

// Kernel that processes all combinations with built-in batching
__global__ void processAllCombinationsKernel(
    int8_t* dataA, int* offsetsA, int* sizesA, int numItemsA,
    int8_t* dataB, int* offsetsB, int* sizesB, int numItemsB,
    int threshold, int level,
    int* resultData, int* resultSizes, int* validFlags, int maxResultSize,
    int combinationsPerThread
) {
    // Calculate global thread ID
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Each thread processes multiple combinations using grid-stride loop
    for (int i = 0; i < combinationsPerThread; i++) {
        // Calculate combination index for this thread and iteration
        int combinationIdx = threadId * combinationsPerThread + i;
        
        // Check if this combination index is valid
        if (combinationIdx >= numItemsA * numItemsB) {
            return;
        }
        
        // Calculate setA and setB indices from the combination index
        int idxA = combinationIdx / numItemsB;
        int idxB = combinationIdx % numItemsB;
        
        // Get vectors from set A and set B
        int offsetA = offsetsA[idxA];
        int sizeA = sizesA[idxA];
        int offsetB = offsetsB[idxB];
        int sizeB = sizesB[idxB];
        
        // Local working memory for unique elements
        int localSet[MAX_ELEMENTS_PER_VECTOR * 2];
        int localSetSize = 0;
        
        // Merge vectors, keeping only unique elements
        for (int j = 0; j < sizeA; j++) {
            int val = dataA[offsetA + j];
            if (!deviceContains(localSet, localSetSize, val)) {
                localSet[localSetSize++] = val;
            }
        }
        
        for (int j = 0; j < sizeB; j++) {
            int val = dataB[offsetB + j];
            if (!deviceContains(localSet, localSetSize, val)) {
                localSet[localSetSize++] = val;
            }
        }
        
        // Check threshold condition
        bool isValid = (threshold == 0 || localSetSize <= threshold);
        
        // If valid, copy result to output buffer
        if (isValid) {
            validFlags[combinationIdx] = 1;
            resultSizes[combinationIdx] = localSetSize;
            
            int resultOffset = combinationIdx * maxResultSize;
            for (int j = 0; j < localSetSize; j++) {
                resultData[resultOffset + j] = localSet[j];
            }
        } else {
            validFlags[combinationIdx] = 0;
            resultSizes[combinationIdx] = localSetSize; // Store size for debugging
        }
    }
}

//-------------------------------------------------------------------------
// Core processing functions that match Python exactly
//-------------------------------------------------------------------------

// Represents an item in a processing level of the tree fold
struct LevelItem {
    CudaSet set;
    std::string streamPath;
    int numItems;
    int id;
    bool needsCleanup; // True if this is an intermediate result that should be freed/deleted

    bool isStreamed() const { return !streamPath.empty(); }
};

// Global counter for unique item IDs
static int levelItemCounter = 0;

// Helper to get the first vector from a CudaSet for threshold calculation
std::vector<int> getFirstVectorFromCudaSet(const CudaSet& set) {
    if (set.numItems == 0) return {};
    int size;
    CHECK_CUDA_ERROR(hipMemcpy(&size, set.sizes, sizeof(int), hipMemcpyDeviceToHost));
    
    std::vector<int8_t> h_firstVector8(size);
    int offset = 0; // First vector is always at offset 0
    CHECK_CUDA_ERROR(hipMemcpy(h_firstVector8.data(), set.data + offset, size * sizeof(int8_t), hipMemcpyDeviceToHost));
    
    std::vector<int> firstVector(size);
    for(int i = 0; i < size; ++i) firstVector[i] = h_firstVector8[i];
    return firstVector;
}

// Helper to get the first vector from a streamed file
std::vector<int> getFirstVectorFromStream(const std::string& filePath) {
    FILE* inFile = fopen(filePath.c_str(), "rb");
    if (!inFile) return {};

    int vecSize = 0;
    size_t elementsRead = fread(&vecSize, sizeof(int), 1, inFile);
    if (elementsRead == 0) {
        fclose(inFile);
        return {};
    }

    std::vector<int> firstVec(vecSize);
    fread(firstVec.data(), sizeof(int), vecSize, inFile);
    fclose(inFile);
    return firstVec;
}

// Modified threshold computation to handle streamed and in-memory sets
int computeThreshold(const LevelItem& itemA, const LevelItem& itemB) {
    if (itemA.numItems == 0 || itemB.numItems == 0) return 0;

    // Get the first vector from item A
    std::vector<int> firstVectorA = itemA.isStreamed() ? 
        getFirstVectorFromStream(itemA.streamPath) : 
        getFirstVectorFromCudaSet(itemA.set);

    // Get the first vector from item B
    std::vector<int> firstVectorB = itemB.isStreamed() ? 
        getFirstVectorFromStream(itemB.streamPath) : 
        getFirstVectorFromCudaSet(itemB.set);

    if (firstVectorA.empty() || firstVectorB.empty()) return 0;
    
    // The rest of the logic is the same: find unique absolute values
    std::set<int> uniqueAbsValues;
    for (int value : firstVectorA) uniqueAbsValues.insert(abs(value));
    for (int value : firstVectorB) uniqueAbsValues.insert(abs(value));
        
    return uniqueAbsValues.size();
}

// Helper function to extract a subset from a CudaSet
CudaSet extractSubset(const CudaSet& set, int startIndex, int count, bool verbose) {
    if (count <= 0) {
        // Return empty set
        CudaSet emptySet;
        emptySet.numItems = 0;
        emptySet.totalElements = 0;
        emptySet.data = nullptr;
        emptySet.offsets = nullptr;
        emptySet.sizes = nullptr;
        emptySet.deviceBuffer = nullptr;
        emptySet.bufferSize = 0;
        return emptySet;
    }
    
    // Copy size and offset information for the slice
    std::vector<int> hostSizes(count);
    std::vector<int> hostOffsets(count);
    
    CHECK_CUDA_ERROR(hipMemcpy(hostSizes.data(), set.sizes + startIndex, 
                              count * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(hostOffsets.data(), set.offsets + startIndex, 
                              count * sizeof(int), hipMemcpyDeviceToHost));
    
    // Calculate total elements in the subset
    int totalElements = 0;
    for (int i = 0; i < count; i++) {
        totalElements += hostSizes[i];
    }
    
    // Allocate memory for the subset
    CudaSet subSet = allocateCudaSet(count, totalElements);
    
    // Copy offset and size information
    std::vector<int> newOffsets(count);
    int currentOffset = 0;
    for (int i = 0; i < count; i++) {
        newOffsets[i] = currentOffset;
        currentOffset += hostSizes[i];
    }
    
    CHECK_CUDA_ERROR(hipMemcpy(subSet.sizes, hostSizes.data(), 
                              count * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(subSet.offsets, newOffsets.data(), 
                              count * sizeof(int), hipMemcpyHostToDevice));
    
    // Copy data elements for each vector
    for (int i = 0; i < count; i++) {
        int srcOffset = hostOffsets[i];
        int dstOffset = newOffsets[i];
        int size = hostSizes[i];
        
        CHECK_CUDA_ERROR(hipMemcpy(subSet.data + dstOffset, set.data + srcOffset, 
                                  size * sizeof(int8_t), hipMemcpyDeviceToDevice));
    }
    
    return subSet;
}

// Flushes a batch of results to disk to keep host RAM usage low, using Zstandard compression.
// Each flush operation writes one or more compressed chunks to the file.
size_t flushResultsToDisk(std::unordered_map<size_t, std::vector<int>>& results, 
                        const char* outputPath, bool& isFirstWrite, bool verbose) {
    if (results.empty()) {
        return 0;
    }

    size_t itemsToFlush = results.size();
    if (verbose) {
        printf("    Flushing %zu items to %s (append: %s)\n", itemsToFlush, outputPath, isFirstWrite ? "false" : "true");
    }

    FILE* outFile = fopen(outputPath, isFirstWrite ? "wb" : "ab");
    if (!outFile) {
        fprintf(stderr, "Error: Could not open output file %s for appending\n", outputPath);
        return 0;
    }

    if (isFirstWrite) {
        isFirstWrite = false;
    }

    const int BATCH_SIZE = 50000;
    std::vector<std::vector<int>> batchVectors;
    batchVectors.reserve(BATCH_SIZE);

    auto it = results.begin();
    while (it != results.end()) {
        // Collect a batch of vectors
        for (int i = 0; i < BATCH_SIZE && it != results.end(); ++i) {
            batchVectors.push_back(std::move(it->second));
            ++it;
        }

        if (batchVectors.empty()) continue;

        // Sort the batch lexicographically before serialization
        std::sort(std::execution::par, batchVectors.begin(), batchVectors.end());

        // Serialize the batch
        std::vector<char> serializedBatch;
        size_t totalElements = 0;
        for(const auto& vec : batchVectors) totalElements += vec.size();
        serializedBatch.reserve(sizeof(size_t) + batchVectors.size() * sizeof(int) + totalElements * sizeof(int));
        
        size_t numVectorsInBatch = batchVectors.size();
        serializedBatch.insert(serializedBatch.end(), (char*)&numVectorsInBatch, (char*)&numVectorsInBatch + sizeof(size_t));

        for (const auto& vec : batchVectors) {
            int vecSize = vec.size();
            serializedBatch.insert(serializedBatch.end(), (char*)&vecSize, (char*)&vecSize + sizeof(int));
            serializedBatch.insert(serializedBatch.end(), (char*)vec.data(), (char*)vec.data() + vecSize * sizeof(int));
        }

        // Compress the serialized batch
        size_t const cBuffSize = ZSTD_compressBound(serializedBatch.size());
        std::vector<char> compressedBatch(cBuffSize);
        size_t const cSize = ZSTD_compress(compressedBatch.data(), cBuffSize, serializedBatch.data(), serializedBatch.size(), 1);
        
        if (ZSTD_isError(cSize)) {
            fprintf(stderr, "ZSTD compression error: %s\n", ZSTD_getErrorName(cSize));
            fclose(outFile);
            return 0;
        }

        // Write the compressed chunk to disk: [size][data]
        uint64_t compressedSize = cSize;
        fwrite(&compressedSize, sizeof(uint64_t), 1, outFile);
        fwrite(compressedBatch.data(), 1, compressedSize, outFile);

        batchVectors.clear();
    }

    fclose(outFile);

    // CRITICAL: Clear the map to free host RAM
    results.clear();
    
    // Force the hash map to release its memory by swapping with an empty map
    std::unordered_map<size_t, std::vector<int>>().swap(results);
    
    // Force allocator to return memory to OS
    malloc_trim(0);
    
    if (verbose) {
        printf("    Flush complete. RAM freed.\n");
    }
    return itemsToFlush;
}

// Loads a compressed chunk of vectors from a binary file into a CudaSet
CudaSet loadCudaSetChunkFromBinary(const char* filePath, long long& fileOffset, int maxVectorsToLoad, bool verbose) {
    // Open file in binary read mode
    FILE* inFile = fopen(filePath, "rb");
    if (!inFile) {
        if (verbose) printf("    Warning: Could not open file for chunk loading: %s\n", filePath);
        CudaSet emptySet = {nullptr, nullptr, nullptr, 0, 0, nullptr, 0};
        return emptySet;
    }

    // Seek to the starting offset
    fseek(inFile, fileOffset, SEEK_SET);

    // Read the size of the next compressed chunk
    uint64_t compressedSize = 0;
    size_t elementsRead = fread(&compressedSize, sizeof(uint64_t), 1, inFile);
    if (elementsRead == 0) {
        // End of file
        fclose(inFile);
        CudaSet emptySet = {nullptr, nullptr, nullptr, 0, 0, nullptr, 0};
        return emptySet;
    }

    // Read the compressed chunk
    std::vector<char> compressedChunk(compressedSize);
    fread(compressedChunk.data(), 1, compressedSize, inFile);

    // Decompress the chunk
    unsigned long long const rSize = ZSTD_getFrameContentSize(compressedChunk.data(), compressedSize);
    if (rSize == ZSTD_CONTENTSIZE_ERROR || rSize == ZSTD_CONTENTSIZE_UNKNOWN) {
        fprintf(stderr, "Error: ZSTD cannot get decompressed size of the frame\n");
        fclose(inFile);
        return {nullptr, nullptr, nullptr, 0, 0, nullptr, 0};
    }
    
    std::vector<char> decompressedChunk(rSize);
    size_t const dSize = ZSTD_decompress(decompressedChunk.data(), rSize, compressedChunk.data(), compressedSize);

    if (ZSTD_isError(dSize) || dSize != rSize) {
        fprintf(stderr, "ZSTD decompression error: %s\n", ZSTD_getErrorName(dSize));
        fclose(inFile);
        return {nullptr, nullptr, nullptr, 0, 0, nullptr, 0};
    }

    // Deserialize the vectors from the decompressed chunk
    HostSet hostSet;
    char* bufferPtr = decompressedChunk.data();
    
    size_t numVectorsInChunk = *(size_t*)bufferPtr;
    bufferPtr += sizeof(size_t);
    
    hostSet.vectors.reserve(numVectorsInChunk);

    for (size_t i = 0; i < numVectorsInChunk; ++i) {
        int vecSize = *(int*)bufferPtr;
        bufferPtr += sizeof(int);
        
        std::vector<int> tempVec(vecSize);
        memcpy(tempVec.data(), bufferPtr, vecSize * sizeof(int));
        bufferPtr += vecSize * sizeof(int);

        hostSet.vectors.push_back(std::move(tempVec));
    }

    // Update the file offset for the next call
    fileOffset = ftell(inFile);
    fclose(inFile);

    // If no vectors were loaded, return an empty set
    if (hostSet.vectors.empty()) {
        CudaSet emptySet = {nullptr, nullptr, nullptr, 0, 0, nullptr, 0};
        return emptySet;
    }

    if (verbose) {
        printf("    Loaded and decompressed chunk of %zu vectors from %s\n", hostSet.vectors.size(), filePath);
    }

    // Convert the host set to a CudaSet and return
    CudaSet cudaSet;
    copyHostToDevice(hostSet, &cudaSet);
    return cudaSet;
}

// Global counter for unique stream file names
static int streamFileCounter = 0;

// Helper function to get current memory usage
size_t getCurrentMemoryUsage() {
    std::ifstream status("/proc/self/status");
    std::string line;
    while (std::getline(status, line)) {
        if (line.substr(0, 6) == "VmRSS:") {
            std::istringstream iss(line);
            std::string label, value, unit;
            iss >> label >> value >> unit;
            return std::stoull(value) * 1024; // Convert KB to bytes
        }
    }
    return 0;
}

// An internal version of processPair that is guaranteed to run on the GPU without triggering another streaming operation.
// It also contains the full deduplication logic.
ProcessResult processPair_inMemory(const CudaSet& setA, const CudaSet& setB, int threshold, int level, bool verbose) {
    long long totalCombinations = (long long)setA.numItems * (long long)setB.numItems;
    
    // Calculate buffer size needed
    int maxResultsPerThread = 4;
    int threadsNeeded = (totalCombinations + maxResultsPerThread - 1) / maxResultsPerThread;
    
    // Determine thread block configuration
    int threadsPerBlock = 256;
    int blocksNeeded = (threadsNeeded + threadsPerBlock - 1) / threadsPerBlock;
    
    // Limit blocks to avoid excessive memory usage
    const int MAX_BLOCKS = 16384;
    if (blocksNeeded > MAX_BLOCKS) {
        blocksNeeded = MAX_BLOCKS;
        maxResultsPerThread = (totalCombinations + (blocksNeeded * threadsPerBlock) - 1) / (blocksNeeded * threadsPerBlock);
    }
    
    // Allocate result buffer
    CombinationResultBuffer resultBuffer = allocateCombinationResultBuffer(setA.numItems, setB.numItems, MAX_ELEMENTS_PER_VECTOR);
    
    // Launch kernel
    processAllCombinationsKernel<<<blocksNeeded, threadsPerBlock>>>(
        setA.data, setA.offsets, setA.sizes, setA.numItems,
        setB.data, setB.offsets, setB.sizes, setB.numItems,
        threshold, level,
        resultBuffer.data, resultBuffer.sizes, resultBuffer.validFlags, resultBuffer.maxResultSize,
        maxResultsPerThread
    );
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    // Count valid combinations
    std::vector<int> hostValidFlags(resultBuffer.numCombinations);
    CHECK_CUDA_ERROR(hipMemcpy(hostValidFlags.data(), resultBuffer.validFlags, 
                              resultBuffer.numCombinations * sizeof(int), hipMemcpyDeviceToHost));
    
    int validCount = 0;
    for (int i = 0; i < resultBuffer.numCombinations; i++) {
        if (hostValidFlags[i]) validCount++;
    }

    if (verbose) {
        printf("    Found %d valid combinations out of %lld total\n", validCount, totalCombinations);
    }

    std::vector<std::vector<int>> validCombinations;
	if (validCount > 0) {
        if (verbose) {
		    printf("    Copying result data for %d valid combinations...\n", validCount);
		}

        std::vector<int> hostSizes(resultBuffer.numCombinations);
        CHECK_CUDA_ERROR(hipMemcpy(hostSizes.data(), resultBuffer.sizes, 
                                  resultBuffer.numCombinations * sizeof(int), hipMemcpyDeviceToHost));
		
		std::vector<int> hostResultData(resultBuffer.numCombinations * resultBuffer.maxResultSize);
		CHECK_CUDA_ERROR(hipMemcpy(hostResultData.data(), resultBuffer.data,
		                          resultBuffer.numCombinations * resultBuffer.maxResultSize * sizeof(int),
		                          hipMemcpyDeviceToHost));
		
        // Progress reporting variables
		int reportInterval = validCount > 1000 ? validCount / 10 : validCount;
		int lastReportedCount = 0;
		int collectedCount = 0;

		for (int i = 0; i < resultBuffer.numCombinations; i++) {
		    if (hostValidFlags[i]) {
		        int size = hostSizes[i];
		        std::vector<int> combination(size);
		        int offset = i * resultBuffer.maxResultSize;
		        for (int j = 0; j < size; j++) {
		            combination[j] = hostResultData[offset + j];
		        }
		        validCombinations.push_back(combination);
                collectedCount++;
                
                // Progress reporting for large result sets
		        if (verbose && validCount > 1000 && collectedCount - lastReportedCount >= reportInterval) {
		            printf("    Collected %d of %d valid combinations (%.1f%%)\n", 
		                   collectedCount, validCount, 100.0 * collectedCount / validCount);
		            lastReportedCount = collectedCount;
		        }
		    }
		}

        if (verbose && validCount > 1000) {
		    printf("    Collection complete: %d combinations collected\n", collectedCount);
		}
	}

	freeCombinationResultBuffer(&resultBuffer);

    // Remove duplicates
    if (validCombinations.size() > 1) {
        if (verbose) {
            printf("    Deduplicating %zu combinations...\n", validCombinations.size());
        }
        for (auto& combination : validCombinations) {
            std::sort(combination.begin(), combination.end());
        }
        std::sort(std::execution::par, validCombinations.begin(), validCombinations.end());
        validCombinations.erase(std::unique(validCombinations.begin(), validCombinations.end()), validCombinations.end());
        if (verbose) {
            printf("    Deduplication complete: %zu unique combinations.\n", validCombinations.size());
        }
    }

    // Create result set
    HostSet resultHostSet;
    resultHostSet.vectors = validCombinations;
    
    CudaSet resultCudaSet;
    copyHostToDevice(resultHostSet, &resultCudaSet);
    
    return {resultCudaSet, ""};
}


ProcessResult processPair(const CudaSet& setA, const CudaSet& setB, int threshold, int level, bool verbose, bool allowStreaming = true) {
    int numItemsA = setA.numItems;
    int numItemsB = setB.numItems;
    
    if (verbose) {
        printf("  Processing pair at level %d: Set A (%d items) + Set B (%d items), threshold = %d\n", 
               level, numItemsA, numItemsB, threshold);
    }
    
    // Empty result for empty inputs
    if (numItemsA == 0 || numItemsB == 0) {
        CudaSet emptySet = {nullptr, nullptr, nullptr, 0, 0, nullptr, 0};
        return {emptySet, ""};
    }

    // For extremely large combinations, use the memory-efficient approach (if allowed)
    long long totalCombinations = (long long)numItemsA * (long long)numItemsB;
    if (allowStreaming && totalCombinations > 3000000LL) { // 3 million threshold
        char streamFilePath[256];
        sprintf(streamFilePath, "zdd_stream_level%d_file%d.bin", level, streamFileCounter++);
        
        // This is now a two-step process: compute, then write.
        std::unordered_map<size_t, std::vector<int>> results;
        processLargePair(setA, setB, threshold, level, verbose, results);
        bool isFirstWrite = true;
        flushResultsToDisk(results, streamFilePath, isFirstWrite, verbose);
        
        return { {nullptr, nullptr, nullptr, 0, 0, nullptr, 0}, streamFilePath};
    }
    
    // All other cases are processed in-memory.
    if (verbose) {
        printf("    Using in-memory GPU processing.\n");
    }
    return processPair_inMemory(setA, setB, threshold, level, verbose);
}


void processLargePair(const CudaSet& setA, const CudaSet& setB, int threshold, int level, bool verbose, 
                     std::unordered_map<size_t, std::vector<int>>& uniqueResults,
                     std::function<void()> flushCallback) {
    int numItemsA = setA.numItems;
    int numItemsB = setB.numItems;
    long long totalCombinations = (long long)numItemsA * (long long)numItemsB;
    
    if (verbose) {
        printf("  Processing large pair with tiled approach: Set A (%d items) + Set B (%d items), threshold = %d\n", 
               numItemsA, numItemsB, threshold);
        printf("  Total combinations: %lld - using tiled processing\n", totalCombinations);
    }
    
    // Calculate number of tiles
    int numTilesA = (numItemsA + TILE_SIZE_A - 1) / TILE_SIZE_A;
    int numTilesB = (numItemsB + TILE_SIZE_B - 1) / TILE_SIZE_B;
    int totalTiles = numTilesA * numTilesB;
    
    if (verbose) {
        printf("    Processing in %d x %d = %d tiles\n", numTilesA, numTilesB, totalTiles);
        printf("    Tile dimensions: %d x %d items\n", TILE_SIZE_A, TILE_SIZE_B);
    }
    
    // Hash function for vectors
    auto hashVector = [](const std::vector<int>& vec) {
        size_t hash = vec.size();
        for (int val : vec) {
            hash ^= std::hash<int>{}(val) + 0x9e3779b9 + (hash << 6) + (hash >> 2);
        }
        return hash;
    };
    
    // Process each tile
    int tilesProcessed = 0;
    int lastProgressUpdate = 0;
    
    for (int tileA = 0; tileA < numTilesA; tileA++) {
        int startA = tileA * TILE_SIZE_A;
        int endA = std::min(startA + TILE_SIZE_A, numItemsA);
        int sizeA = endA - startA;
        
        // Create a sub-set for this tile of setA
        CudaSet tileSetA = extractSubset(setA, startA, sizeA, false);
        
        for (int tileB = 0; tileB < numTilesB; tileB++) {
            int startB = tileB * TILE_SIZE_B;
            int endB = std::min(startB + TILE_SIZE_B, numItemsB);
            int sizeB = endB - startB;
            
            tilesProcessed++;
            
            // Update progress periodically
            int progressPercentage = (tilesProcessed * 100) / totalTiles;
            if (verbose && (progressPercentage > lastProgressUpdate || tilesProcessed == totalTiles)) {
                printf("      Processing tile [%d,%d] x [%d,%d] (Tile %d of %d - %d%% complete)\n", 
                      startA, endA-1, startB, endB-1, tilesProcessed, totalTiles, progressPercentage);
                lastProgressUpdate = progressPercentage;
            }
            
            // Create a sub-set for this tile of setB
            CudaSet tileSetB = extractSubset(setB, startB, sizeB, false);
            
            // Process this tile pair directly
            int numTileItemsA = tileSetA.numItems;
            int numTileItemsB = tileSetB.numItems;
            
            if (numTileItemsA == 0 || numTileItemsB == 0) {
                freeCudaSet(&tileSetB);
                continue;
            }
            
            ProcessResult chunkResult = processPair_inMemory(tileSetA, tileSetB, threshold, level, false);

            if (chunkResult.set.numItems > 0) {
                HostSet hostResult = copyDeviceToHost(chunkResult.set);
                for (auto& vec : hostResult.vectors) {
                    std::sort(vec.begin(), vec.end());
                    size_t hash = hashVector(vec);
                    if (uniqueResults.find(hash) == uniqueResults.end()) {
                        uniqueResults[hash] = std::move(vec);
                    }
                }
                freeCudaSet(&chunkResult.set);
            }
            
            // Free tile resources
            freeCudaSet(&tileSetB);
            
            // Check if we need to flush results periodically during tiled processing
            if (flushCallback && uniqueResults.size() > RESULTS_FLUSH_THRESHOLD) {
                if (verbose) {
                    printf("      Flushing results during tiled processing (%zu items)...\n", uniqueResults.size());
                }
                flushCallback();
                
                // Force GPU memory cleanup after flush
                CHECK_CUDA_ERROR(hipDeviceSynchronize());
                CHECK_CUDA_ERROR(hipMemGetInfo(nullptr, nullptr)); // Force memory manager update
                
                // Additional aggressive cleanup
                if (verbose) {
                    printf("      Post-flush cleanup: uniqueResults size = %zu\n", uniqueResults.size());
                }
            }
            
            // More frequent flushing for very large tile results
            if (flushCallback && uniqueResults.size() > RESULTS_FLUSH_THRESHOLD / 4) {
                if (verbose) {
                    printf("      Early flush triggered at %zu items\n", uniqueResults.size());
                }
                flushCallback();
                CHECK_CUDA_ERROR(hipDeviceSynchronize());
            }
        }
        
        // Free tile resources
        freeCudaSet(&tileSetA);
    }
    
    if (verbose) {
        printf("  Tiled processing complete. Results added to shared collection.\n");
    }
}
// Special handling for converting a set to unique elements (for level 1 carry-over)
CudaSet convertSetToUnique(const CudaSet& set, bool verbose) {
    int numItems = set.numItems;
    
    // Allocate host vectors 
    std::vector<int> hostOffsets(numItems);
    std::vector<int> hostSizes(numItems);
    
    CHECK_CUDA_ERROR(hipMemcpy(hostSizes.data(), set.sizes, numItems * sizeof(int), hipMemcpyDeviceToHost));
    
    // Calculate max possible size for outputs
    int totalOutputSize = 0;
    for (int i = 0; i < numItems; i++) {
        totalOutputSize += hostSizes[i]; // Worst case: all elements are unique
    }
    
    // Create output arrays
    int8_t* d_outputData = nullptr;
    int* d_outputOffsets = nullptr;
    int* d_outputSizes = nullptr;
    
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_outputData, totalOutputSize * sizeof(int8_t)));
    CHECK_CUDA_ERROR(hipMalloc(&d_outputOffsets, numItems * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_outputSizes, numItems * sizeof(int)));
    
    // Calculate output offsets (equivalent to the input offsets)
    CHECK_CUDA_ERROR(hipMemcpy(d_outputOffsets, set.offsets, numItems * sizeof(int), hipMemcpyDeviceToDevice));
    
    // Launch parallel kernel
    int threadsPerBlock = 256;
    int blocks = (numItems + threadsPerBlock - 1) / threadsPerBlock;
    
    convertToUniqueKernel<<<blocks, threadsPerBlock>>>(
        set.data, set.offsets, set.sizes, numItems,
        d_outputData, d_outputOffsets, d_outputSizes, MAX_ELEMENTS_PER_VECTOR
    );
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    // Create result set
    CudaSet resultSet;
    resultSet.data = d_outputData;
    resultSet.offsets = d_outputOffsets;
    resultSet.sizes = d_outputSizes;
    resultSet.numItems = numItems;
    resultSet.totalElements = totalOutputSize;
    resultSet.deviceBuffer = nullptr;
    resultSet.bufferSize = 0;
    
    if (verbose) {
        printf("  Converting carried-over set for level 2\n");
        printf("  Carried over the last set with %d items\n", numItems);
    }
    
    return resultSet;
}

// Core function to process a pair where at least one input is streamed to disk
LevelItem processStreamedPair(LevelItem& itemA, LevelItem& itemB, int threshold, int level, bool verbose) {
    char outStreamPath[256];
    sprintf(outStreamPath, "zdd_stream_level%d_file%d.bin", level, streamFileCounter++);
    
    if (verbose) {
        printf("  Processing streamed pair -> %s\n", outStreamPath);
        printf("    Item A (ID %d): %s (%d items)\n", itemA.id, itemA.isStreamed() ? itemA.streamPath.c_str() : "in-memory", itemA.numItems);
        printf("    Item B (ID %d): %s (%d items)\n", itemB.id, itemB.isStreamed() ? itemB.streamPath.c_str() : "in-memory", itemB.numItems);
    }

    std::unordered_map<size_t, std::vector<int>> uniqueResults;
    bool isFirstWrite = true;
    size_t totalItemsWritten = 0;
    
    auto hashVector = [](const std::vector<int>& vec) {
        size_t hash = vec.size();
        for (int val : vec) hash ^= std::hash<int>{}(val) + 0x9e3779b9 + (hash << 6) + (hash >> 2);
        return hash;
    };

    long long offsetA = 0;
    while(offsetA < itemA.numItems) {
        CudaSet chunkA;
        long long currentChunkSizeA = std::min((long long)CHUNK_SIZE, (long long)itemA.numItems - offsetA);

        if (itemA.isStreamed()) {
            chunkA = loadCudaSetChunkFromBinary(itemA.streamPath.c_str(), offsetA, currentChunkSizeA, false);
        } else {
            chunkA = extractSubset(itemA.set, offsetA, currentChunkSizeA, false);
        }

        if(verbose) printf("    Loaded chunk A (items %lld-%lld) with %d items.\n", offsetA, offsetA + chunkA.numItems -1, chunkA.numItems);
        if (chunkA.numItems == 0) {
            offsetA += currentChunkSizeA;
            continue;
        }

        long long offsetB = 0;
        while(offsetB < itemB.numItems) {
            CudaSet chunkB;
            long long currentChunkSizeB = std::min((long long)CHUNK_SIZE, (long long)itemB.numItems - offsetB);
            
            if (itemB.isStreamed()) {
                chunkB = loadCudaSetChunkFromBinary(itemB.streamPath.c_str(), offsetB, currentChunkSizeB, false);
            } else {
                chunkB = extractSubset(itemB.set, offsetB, currentChunkSizeB, false);
            }
            
            if(verbose) printf("      Loaded chunk B (items %lld-%lld) with %d items.\n", offsetB, offsetB + chunkB.numItems - 1, chunkB.numItems);
            if (chunkB.numItems == 0) {
                offsetB += currentChunkSizeB;
                continue;
            }
            
            if(verbose) printf("      Processing chunk pair with tiled approach.\n");
            
            // Create a flush callback that will be called during tiled processing
            auto flushCallback = [&]() {
                size_t memBefore = getCurrentMemoryUsage();
                if(verbose) printf("    Flushing intermediate results to disk (%zu items, RAM: %.1f MB)...\n", 
                                 uniqueResults.size(), memBefore / 1024.0 / 1024.0);
                totalItemsWritten += flushResultsToDisk(uniqueResults, outStreamPath, isFirstWrite, verbose);
                
                std::unordered_map<size_t, std::vector<int>>().swap(uniqueResults);
                malloc_trim(0);
                
                size_t memAfter = getCurrentMemoryUsage();
                long long memDiff = (long long)memBefore - (long long)memAfter;
                if(verbose) printf("    Post-flush RAM: %.1f MB (change: %+.1f MB)\n", 
                                 memAfter / 1024.0 / 1024.0, memDiff / 1024.0 / 1024.0);
            };
            
            processLargePair(chunkA, chunkB, threshold, level, false, uniqueResults, flushCallback);

            if (uniqueResults.size() > RESULTS_FLUSH_THRESHOLD) {
                if(verbose) printf("    Flushing intermediate results to disk (%zu items)...\n", uniqueResults.size());
                totalItemsWritten += flushResultsToDisk(uniqueResults, outStreamPath, isFirstWrite, verbose);
                
                // Force memory cleanup after flush
                CHECK_CUDA_ERROR(hipDeviceSynchronize());
                CHECK_CUDA_ERROR(hipMemGetInfo(nullptr, nullptr)); // Force memory manager update
                malloc_trim(0);
            }
            
            freeCudaSet(&chunkB);
            offsetB += currentChunkSizeB;
        }
        
        freeCudaSet(&chunkA);
        offsetA += currentChunkSizeA;
    }
    
    // Final flush for any remaining results
    if (!uniqueResults.empty()) {
        if (verbose) printf("    Flushing final results to disk (%zu items)...\n", uniqueResults.size());
        totalItemsWritten += flushResultsToDisk(uniqueResults, outStreamPath, isFirstWrite, verbose);
    }
    
    if (verbose) printf("  --> Streamed pair processing complete. Result: %zu items in %s\n", totalItemsWritten, outStreamPath);

    return { {}, outStreamPath, (int)totalItemsWritten, levelItemCounter++, true };
}

// Tree fold operations (maintains sequential dependencies but optimizes within each step)
LevelItem treeFoldOperations(const std::vector<CudaSet>& sets, bool verbose) {
    if (sets.empty()) {
        return { {nullptr, nullptr, nullptr, 0, 0, nullptr, 0}, "", 0, -1, false };
    }

    std::vector<std::string> tempFiles; // Keep track of intermediate files to delete

    // Initialize the first level with LevelItems
    std::vector<LevelItem> currentLevel;
    for (const auto& s : sets) {
        currentLevel.push_back({s, "", s.numItems, levelItemCounter++, false});
    }

    if (currentLevel.size() == 1) {
        return currentLevel[0];
    }
    
    if (verbose) {
        printf("Starting tree-fold operations with %zu sets\n", sets.size());
        for (const auto& item : currentLevel) {
            printf("  Set %d: %d items\n", item.id, item.numItems);
        }
    }
    
    int level = 0;
    while (currentLevel.size() > 1) {
        level++;
        if (verbose) {
            printf("\nProcessing Level %d with %zu sets\n", level, currentLevel.size());
        }
        
        std::vector<LevelItem> nextLevel;
        std::vector<bool> processed(currentLevel.size(), false);
        
        // --- Phase 1: Process all possible in-memory pairs ---
        bool inMemoryPairFound;
        do {
            inMemoryPairFound = false;
            int bestI = -1, bestJ = -1;
            int lowestThreshold = INT_MAX;

            // Find the best in-memory pair among all non-processed items
            for (size_t i = 0; i < currentLevel.size(); i++) {
                if (processed[i]) continue;
                for (size_t j = i + 1; j < currentLevel.size(); j++) {
                    if (processed[j]) continue;
                    
                    bool willStream = currentLevel[i].isStreamed() || currentLevel[j].isStreamed() ||
                                      (long long)currentLevel[i].numItems * (long long)currentLevel[j].numItems >= 3000000LL;

                    if (!willStream) {
                        int threshold = computeThreshold(currentLevel[i], currentLevel[j]);
                        if (threshold < lowestThreshold) {
                            lowestThreshold = threshold;
                            bestI = i;
                            bestJ = j;
                        }
                    }
                }
            }

            if (bestI != -1) {
                inMemoryPairFound = true;
                processed[bestI] = true;
                processed[bestJ] = true;

                LevelItem& itemA = currentLevel[bestI];
                LevelItem& itemB = currentLevel[bestJ];

                if (verbose) {
                    printf("  --> Selected in-memory pair: Set %d (%d items) + Set %d (%d items) with threshold %d\n", 
                           itemA.id, itemA.numItems, itemB.id, itemB.numItems, lowestThreshold);
                    printf("      Processing pair in-memory (GPU).\n");
                }

                ProcessResult res = processPair(itemA.set, itemB.set, lowestThreshold, level, verbose, true);
                LevelItem resultItem = { res.set, res.streamPath, res.set.numItems, levelItemCounter++, true };
                if (!res.streamPath.empty()) {
                   tempFiles.push_back(res.streamPath);
                }
                nextLevel.push_back(resultItem);
            }
        } while (inMemoryPairFound);

        // --- Phase 2: Process all remaining items using streaming-optimized strategy ---
        std::vector<int> remainingIndices;
        for(size_t i = 0; i < currentLevel.size(); ++i) {
            if (!processed[i]) {
                remainingIndices.push_back(i);
            }
        }

        if (remainingIndices.size() >= 2) {
            if (verbose) {
                printf("  --> No in-memory pairs left. Applying streaming-optimized pairing to %zu sets.\n", remainingIndices.size());
            }

            std::sort(remainingIndices.begin(), remainingIndices.end(),
                      [&](int a, int b) { return currentLevel[a].numItems < currentLevel[b].numItems; });

            int i = 0;
            int j = remainingIndices.size() - 1;
            while (i < j) {
                int idxA = remainingIndices[i];
                int idxB = remainingIndices[j];
                processed[idxA] = true;
                processed[idxB] = true;

                LevelItem& itemA = currentLevel[idxA];
                LevelItem& itemB = currentLevel[idxB];

                int threshold = computeThreshold(itemA, itemB);
                if (verbose) {
                    printf("  --> Pairing smallest (%s:%d items) with largest (%s:%d items) for streaming.\n",
                           itemA.isStreamed() ? "stream" : "mem", itemA.numItems,
                           itemB.isStreamed() ? "stream" : "mem", itemB.numItems);
                }

                LevelItem resultItem = processStreamedPair(itemA, itemB, threshold, level, verbose);
                tempFiles.push_back(resultItem.streamPath);
                nextLevel.push_back(resultItem);

                i++;
                j--;
            }
        }
        
        // Handle any remaining odd set by carrying it over to the next level
        for(size_t i = 0; i < currentLevel.size(); ++i) {
            if(!processed[i]) {
                LevelItem& carriedItem = currentLevel[i];
                if (verbose) {
                    printf("  --> Carrying over odd set %d (%d items) to next level\n", 
                           carriedItem.id, carriedItem.numItems);
                }

                // For level 1, convert the carried-over set to unique elements, as per original logic.
                // This is a special operation that only happens on the first-level carry-over.
                if (level == 1) {
                   CudaSet convertedSet = convertSetToUnique(carriedItem.set, verbose);
                   // The new item is an intermediate result and will need cleanup
                   nextLevel.push_back({convertedSet, "", convertedSet.numItems, levelItemCounter++, true});
                } else {
                   // For other levels, just move the item to the next level.
                   // It's not a new intermediate result, so it doesn't need cleanup yet.
                   carriedItem.needsCleanup = false;
                   nextLevel.push_back(carriedItem);
                }
            }
        }
        
        // Clean up resources from the completed level that were marked for cleanup.
        for(const auto& item : currentLevel) {
            if(item.needsCleanup) {
                if(item.isStreamed()) {
                     remove(item.streamPath.c_str());
                } else {
                    freeCudaSet(&const_cast<CudaSet&>(item.set));
                }
            }
        }
        
        currentLevel = nextLevel;
    }
    
    LevelItem finalItem = currentLevel.empty() ? LevelItem{} : currentLevel[0];
    
    // Clean up all temporary files except the final result file
    for (const auto& file : tempFiles) {
        if (file != finalItem.streamPath) {
             remove(file.c_str());
        }
    }
    
    return finalItem;
}

// Kernel to filter out negative values and sort
__global__ void filterAndSortKernel(int8_t* data, int* offsets, int* sizes, int numVectors, int maxLen) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numVectors) return;
    
    int offset = offsets[idx];
    int originalSize = sizes[idx];
    
    // Step 1: Filter out negatives
    int newSize = 0;
    for (int i = 0; i < originalSize; i++) {
        int val = data[offset + i];
        if (val >= 0) {
            // Keep only non-negative values
            data[offset + newSize] = val;
            newSize++;
        }
    }
    
    // Update size
    sizes[idx] = newSize;
    
    // Step 2: Sort (simple insertion sort)
    for (int i = 1; i < newSize; i++) {
        int key = data[offset + i];
        int j = i - 1;
        
        while (j >= 0 && data[offset + j] > key) {
            data[offset + j + 1] = data[offset + j];
            j--;
        }
        
        data[offset + j + 1] = key;
    }
}

// Function to post-process on GPU before transferring to host
// Function to post-process on GPU then complete ordering on CPU
std::vector<std::vector<int>> gpuPostProcess(const CudaSet& resultSet, bool verbose) {
    // Step 1: Run GPU kernel to filter and sort all vectors internally
    int threadsPerBlock = 256;
    int blocks = (resultSet.numItems + threadsPerBlock - 1) / threadsPerBlock;
    
    if (verbose) {
        printf("Running GPU post-processing on %d vectors\n", resultSet.numItems);
    }
    
    filterAndSortKernel<<<blocks, threadsPerBlock>>>(
        resultSet.data, resultSet.offsets, resultSet.sizes, 
        resultSet.numItems, MAX_ELEMENTS_PER_VECTOR);
    
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    if (verbose) {
        printf("GPU internal sorting complete, transferring to host for final sorting\n");
    }
    
    // Step 2: Process in batches to avoid memory issues
    const int BATCH_SIZE = 100000;
    int totalVectors = resultSet.numItems;
    int batches = (totalVectors + BATCH_SIZE - 1) / BATCH_SIZE;
    
    std::vector<std::vector<int>> processedResults;
    processedResults.reserve(std::min(totalVectors, 10000000)); // Reserve reasonable amount
    
    for (int batch = 0; batch < batches; batch++) {
        int start = batch * BATCH_SIZE;
        int end = std::min(start + BATCH_SIZE, totalVectors);
        
        if (verbose) {
            printf("Processing batch %d/%d (vectors %d to %d)\n", batch+1, batches, start, end-1);
        }
        
        // Extract subset of the CudaSet
        CudaSet batchSet = extractSubset(resultSet, start, end - start, false);
        
        // Process this batch - already filtered and sorted internally by GPU
        HostSet hostBatch = copyDeviceToHost(batchSet);
        
        // Add to results
        for (const auto& vector : hostBatch.vectors) {
            processedResults.push_back(vector);
        }
        
        // Free batch resources
        freeCudaSet(&batchSet);
        
        // Sort intermediate results if getting too large
        if (processedResults.size() > 1000000) {
            if (verbose) {
                printf("  Performing intermediate sort of %zu results\n", processedResults.size());
            }
            std::sort(processedResults.begin(), processedResults.end());
        }
    }
    
    // Final lexicographical sorting of all vectors
    if (verbose) {
        printf("Performing final lexicographical sort of %zu vectors\n", processedResults.size());
    }
    std::sort(processedResults.begin(), processedResults.end());
    
    return processedResults;
}

// Function to read JSON and generate test sets
std::vector<std::vector<std::vector<int>>> generateTestSetsFromJSON(const std::string& filename) {
    // Read JSON file
    std::ifstream file(filename);
    if (!file.is_open()) {
        printf("Error: Could not open file: %s\n", filename.c_str());
        return {};
    }
    
    // Read file into string
    std::stringstream buffer;
    buffer << file.rdbuf();
    std::string jsonData = buffer.str();
    
    // Parse JSON
    SimpleJsonParser parser(jsonData);
    auto clauses = parser.parseClauses();
    auto matrices = parser.parseMatrices();
    
    // Create a map for easier lookup
    std::map<std::string, SimpleJsonParser::Matrix> matrixMap;
    for (auto& m : matrices) {
        matrixMap[m.key] = m;
    }
    
    std::vector<std::vector<std::vector<int>>> testSets;
    
    // Process each clause
    for (auto& clause : clauses) {
        std::string key = clause.key;
        int condition_id1 = clause.condition_id1;
        int condition_id2 = clause.condition_id2;
        int consequence_id = clause.consequence_id;
        
        // Find corresponding matrix
        if (matrixMap.find(key) == matrixMap.end()) {
            printf("No matrix found for clause: %s\n", key.c_str());
            continue;
        }
        
        auto& matrix = matrixMap[key];
        int rows = matrix.rows;
        int cols = matrix.cols;
        auto& matrixData = matrix.data;
        
        printf("  Using matrix '%s': %d rows x %d cols, data size: %zu\n", 
               key.c_str(), rows, cols, matrixData.size());
        
        std::vector<std::vector<int>> testSet;
        
        // Generate sets based on matrix data
        for (int row = 0; row < rows; row++) {
            std::vector<int> testRow;
            
            // Add condition_id1 (always present)
            testRow.push_back(matrixData[row * cols + 0] == 1 ? condition_id1 : -condition_id1);
            
            // Add remaining elements based on matrix values
            if (condition_id2 != -1) {
                // If condition_id2 exists, add it based on matrix value
                if (cols >= 2) {
                    testRow.push_back(matrixData[row * cols + 1] == 1 ? condition_id2 : -condition_id2);
                }
                // If there's a third column, it's for the consequence
                if (cols >= 3) {
                    testRow.push_back(matrixData[row * cols + 2] == 1 ? consequence_id : -consequence_id);
                }
            } else {
                // If condition_id2 doesn't exist, the second column is for consequence
                if (cols >= 2) {
                    testRow.push_back(matrixData[row * cols + 1] == 1 ? consequence_id : -consequence_id);
                }
            }
            
            testSet.push_back(testRow);
        }
        
        printf("  Generated test set with %zu rows\n", testSet.size());
        testSets.push_back(testSet);
    }
    
    printf("\nGenerated %zu test sets in total\n\n", testSets.size());
    
    // Print verification of the testSets
    printf("=== Verification of Generated Test Sets ===\n");
    for (size_t i = 0; i < testSets.size(); i++) {
        printf("Test Set %zu (clause '%c'):\n", i, 'a' + static_cast<char>(i));
        printf("{\n");
        for (size_t j = 0; j < testSets[i].size(); j++) {
            printf("  {");
            for (size_t k = 0; k < testSets[i][j].size(); k++) {
                printf("%d", testSets[i][j][k]);
                if (k < testSets[i][j].size() - 1) {
                    printf(",");
                }
            }
            printf("}");
            if (j < testSets[i].size() - 1) {
                printf(",");
            }
            printf("\n");
        }
        printf("}\n\n");
    }
    
    return testSets;
}

// Run test cases
void runTestCases() {
    std::vector<std::vector<std::vector<int>>> testSets = 
        generateTestSetsFromJSON("kelsen_data.json");
    
    // Show input sets
    for (size_t i = 0; i < testSets.size(); i++) {
        printf("  Set %zu: [", i + 1);
        for (size_t j = 0; j < testSets[i].size() && j < 2; j++) {
            printf("[");
            for (size_t k = 0; k < testSets[i][j].size(); k++) {
                printf("%d", testSets[i][j][k]);
                if (k < testSets[i][j].size() - 1) printf(", ");
            }
            printf("]");
            if (j < testSets[i].size() - 1) printf(", ");
        }
        if (testSets[i].size() > 2) printf("...");
        printf("] (%zu items)\n", testSets[i].size());
    }
    
    // Create host sets
    std::vector<HostSet> hostSets;
    for (const auto& vectors : testSets) {
        hostSets.push_back(createTestSet(vectors));
    }
    
    // Convert host sets to CUDA sets
    std::vector<CudaSet> cudaSets;
    for (const auto& hostSet : hostSets) {
        CudaSet cudaSet;
        copyHostToDevice(hostSet, &cudaSet);
        cudaSets.push_back(cudaSet);
    }
    
    // Record start time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    // Run tree-fold operations
    LevelItem finalResult = treeFoldOperations(cudaSets, true);
    
    // Record end time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    printf("\nTree-fold completed in %.2f ms. Total items: %d\n", milliseconds, finalResult.numItems);
    
    std::vector<std::vector<int>> finalVectors;

    if (finalResult.isStreamed()) {
        printf("Final result is on disk (%s). Loading for post-processing...\n", finalResult.streamPath.c_str());
        // Load all vectors from the final stream file
        long long offset = 0;
        while(true) {
            CudaSet chunk = loadCudaSetChunkFromBinary(finalResult.streamPath.c_str(), offset, 100000, true);
            if (chunk.numItems == 0) break;
            
            HostSet hostChunk = copyDeviceToHost(chunk);
            finalVectors.insert(finalVectors.end(), std::make_move_iterator(hostChunk.vectors.begin()), std::make_move_iterator(hostChunk.vectors.end()));
            freeCudaSet(&chunk);
        }
        if (!finalResult.streamPath.empty()) {
             remove(finalResult.streamPath.c_str()); // Clean up final stream file
        }
        printf("Loaded %zu vectors from final stream file.\n", finalVectors.size());
        
        // Post-process on CPU, then write to file
        std::vector<std::vector<int>> processedResults;
        processedResults.reserve(finalVectors.size());
        for (auto& vec : finalVectors) {
            std::vector<int> positives;
            for (int val : vec) {
                if (val >= 0) positives.push_back(val);
            }
            std::sort(positives.begin(), positives.end());
            processedResults.push_back(std::move(positives));
        }
        std::sort(std::execution::par, processedResults.begin(), processedResults.end());
        finalVectors = std::move(processedResults);

    } else if (finalResult.numItems > 0) {
        printf("Final result is in memory (%d items). Post-processing on GPU...\n", finalResult.numItems);
        // Normal case - process the results from GPU memory
        finalVectors = gpuPostProcess(finalResult.set, true);
        freeCudaSet(&const_cast<CudaSet&>(finalResult.set));
    } else {
        printf("Final result is empty.\n");
    }
    
    printf("Final processed result contains %zu combinations\n", finalVectors.size());
    
    // Open file for writing
    FILE* outFile = fopen("zdd.bin", "wb");
    if (!outFile) {
        fprintf(stderr, "Error: Could not open zdd.bin for writing\n");
    } else {
        for (const auto& vec : finalVectors) {
            int size = vec.size();
            fwrite(&size, sizeof(int), 1, outFile);
            fwrite(vec.data(), sizeof(int), size, outFile);
        }
        fclose(outFile);
        printf("Results written to zdd.bin\n");
    }
    
    // Clean up original sets
    for (size_t i = 0; i < cudaSets.size(); i++) {
        freeCudaSet(&cudaSets[i]);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

//-------------------------------------------------------------------------
// Main function
//-------------------------------------------------------------------------

int main() {
    // Initialize CUDA
    int deviceCount;
    CHECK_CUDA_ERROR(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA devices found.\n");
        return EXIT_FAILURE;
    }
    CHECK_CUDA_ERROR(hipSetDevice(0));
    
    // Run tests
    runTestCases();
    
    // Clean up
    CHECK_CUDA_ERROR(hipDeviceReset());
    
    return 0;
}
